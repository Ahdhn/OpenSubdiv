
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 512

__global__ void
expand(int src_numrows, int factor,
  int* dst_rows, int* dst_cols, float* dst_vals,
  int* src_rows, int* src_cols, float* src_vals)
{
    int src_row = threadIdx.x + blockIdx.x * blockDim.x;
    if (src_row >= src_numrows)
        return;

    int v_per_row = src_rows[src_row+1] - src_rows[src_row];
    int base = src_rows[src_row];

    for(int src_idx = src_rows[src_row]; src_idx < src_rows[src_row+1]; src_idx++) {
        for(int k = 0; k < factor; k++) {
            int dst_idx = factor*base + k*v_per_row + src_idx-base;
            dst_rows[dst_idx] = factor * src_row + k;
            dst_cols[dst_idx] = factor * src_cols[src_idx] + k;
            dst_vals[dst_idx] = src_vals[src_idx];
        }
    }
}

extern "C" {

void
OsdCusparseExpand(int src_numrows, int factor,
    int* dst_rows, int* dst_cols, float* dst_vals,
    int* src_rows, int* src_cols, float* src_vals)
{
    int blks = (src_numrows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    expand<<<blks,THREADS_PER_BLOCK>>>(src_numrows, factor,
            dst_rows, dst_cols, dst_vals,
            src_rows, src_cols, src_vals);
}

} /* extern C */
